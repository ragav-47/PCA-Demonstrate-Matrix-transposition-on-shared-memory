#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <windows.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
//for __syncthreads()
#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif // !(__CUDACC_RTC__)
#include <hip/device_functions.h>

inline double seconds()
{
    LARGE_INTEGER t, f;
    QueryPerformanceCounter(&t);
    QueryPerformanceFrequency(&f);
    return (double)t.QuadPart / (double)f.QuadPart;
}

# define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

#define BDIMX 16
#define BDIMY 16

void printMatrix(const char* msg, int* matrix, int width, int height)
{
    printf("%s:\n", msg);

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            printf("%4d ", matrix[i * width + j]);
        }
        printf("\n");
    }

    printf("\n");
}

void verifyResults(int* input, int* output, int width, int height)
{
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            if (input[i * width + j] != output[j * height + i])
            {
                printf("Verification failed at position (%d, %d)\n", i, j);
                return;
            }
        }
    }

    printf("Verification successful\n");
}

__global__ void transposeMatrix(int* input, int* output, int width, int height)
{
    __shared__ int tile[BDIMY][BDIMX + 1];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index_in = y * width + x;
        tile[threadIdx.y][threadIdx.x] = input[index_in];
    }

    __syncthreads();

    x = blockIdx.y * blockDim.y + threadIdx.x;
    y = blockIdx.x * blockDim.x + threadIdx.y;

    if (x < height && y < width)
    {
        int index_out = y * height + x;
        output[index_out] = tile[threadIdx.x][threadIdx.y];
    }
}

int main()
{
    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // Set up matrix dimensions
    int width = 256;
    int height = 256;

    size_t size = width * height * sizeof(int);

    // Allocate host memory for input and output matrices
    int* hostInput = (int*)malloc(size);
    int* hostOutput = (int*)malloc(size);

    // Initialize input matrix
    for (int i = 0; i < width * height; i++)
    {
        hostInput[i] = i;
    }

    // Allocate device memory for input and output matrices
    int* deviceInput;
    int* deviceOutput;
    CHECK(hipMalloc((void**)&deviceInput, size));
    CHECK(hipMalloc((void**)&deviceOutput, size));

    // Copy input matrix from host to device
    CHECK(hipMemcpy(deviceInput, hostInput, size, hipMemcpyHostToDevice));

    // Define block and grid dimensions
    dim3 block(BDIMX, BDIMY);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // Perform matrix transpose using shared memory
    CHECK(hipEventRecord(start, 0));
    transposeMatrix << <grid, block >> > (deviceInput, deviceOutput, width, height);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // Copy output matrix from device to host
    CHECK(hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost));

    // Verify the results
    verifyResults(hostInput, hostOutput, width, height);

    // Compute the elapsed time
    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Shared Memory Transpose Time: %.5f ms\n", elapsedTime);

    // Free host and device memory
    free(hostInput);
    free(hostOutput);
    CHECK(hipFree(deviceInput));
    CHECK(hipFree(deviceOutput));

    // Reset device
    CHECK(hipDeviceReset());

    return 0;
}
